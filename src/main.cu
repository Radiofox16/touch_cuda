
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add(int *a, int *b, int *c) {
    *c = *a + *b;
}

int main() {
    int a = 1234, b = 5555, c;

    int *d_a, *d_b, *d_c;
    auto sz = sizeof(int);

    hipMalloc(&d_a, sz);
    hipMalloc(&d_b, sz);
    hipMalloc(&d_c, sz);

    hipMemcpy(d_a, &a, sz, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, sz, hipMemcpyHostToDevice);

    add<<<1, 1>>>(d_a, d_b, d_c);

    hipMemcpy(&c, d_c, sz, hipMemcpyDeviceToHost);

    std::cout << a << " + " << b << " = " << c << std::endl;

    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}